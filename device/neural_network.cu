#include "hip/hip_runtime.h"
#include "neural_network.cuh"
#include "cuda_utils.cuh"
#include <cmath>
#include <vector>

void NeuralNetwork::initWeights(float *weights, int size) {
    float scale = sqrtf(2.0f / size);
    for (int i = 0; i < size; i++) {
        weights[i] = ((float)rand() / RAND_MAX) * scale - (scale / 2.0f);
    }
}

void NeuralNetwork::initBias(float *bias, int size) {
    for (int i = 0; i < size; i++) {
        bias[i] = 0.0f;
    }
}

NeuralNetwork::NeuralNetwork() {
    // Allocate memory on the device
    CHECK_CUDA_CALL(hipMalloc(&weightsInputHidden1, HIDDEN1_SIZE * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&weightsHidden1Hidden2, HIDDEN2_SIZE * HIDDEN1_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&weightsHidden2Output, OUTPUT_SIZE * HIDDEN2_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&biasHidden1, HIDDEN1_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&biasHidden2, HIDDEN2_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&biasOutput, OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA_CALL(hipMalloc(&gradWeightsInputHidden1, HIDDEN1_SIZE * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&gradWeightsHidden1Hidden2, HIDDEN2_SIZE * HIDDEN1_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&gradWeightsHidden2Output, OUTPUT_SIZE * HIDDEN2_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&gradBiasHidden1, HIDDEN1_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&gradBiasHidden2, HIDDEN2_SIZE * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&gradBiasOutput, OUTPUT_SIZE * sizeof(float)));

    // Host memory for initialization
    std::vector<float> hostWeightsInputHidden1(HIDDEN1_SIZE * INPUT_SIZE);
    std::vector<float> hostWeightsHidden1Hidden2(HIDDEN2_SIZE * HIDDEN1_SIZE);
    std::vector<float> hostWeightsHidden2Output(OUTPUT_SIZE * HIDDEN2_SIZE);
    std::vector<float> hostBiasHidden1(HIDDEN1_SIZE);
    std::vector<float> hostBiasHidden2(HIDDEN2_SIZE);
    std::vector<float> hostBiasOutput(OUTPUT_SIZE);

    initWeights(hostWeightsInputHidden1.data(), HIDDEN1_SIZE * INPUT_SIZE);
    initWeights(hostWeightsHidden1Hidden2.data(), HIDDEN2_SIZE * HIDDEN1_SIZE);
    initWeights(hostWeightsHidden2Output.data(), OUTPUT_SIZE * HIDDEN2_SIZE);
    initBias(hostBiasHidden1.data(), HIDDEN1_SIZE);
    initBias(hostBiasHidden2.data(), HIDDEN2_SIZE);
    initBias(hostBiasOutput.data(), OUTPUT_SIZE);

    CHECK_CUDA_CALL(hipMemcpy(weightsInputHidden1, hostWeightsInputHidden1.data(), 
                              HIDDEN1_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(weightsHidden1Hidden2, hostWeightsHidden1Hidden2.data(), 
                              HIDDEN2_SIZE * HIDDEN1_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(weightsHidden2Output, hostWeightsHidden2Output.data(), 
                              OUTPUT_SIZE * HIDDEN2_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(biasHidden1, hostBiasHidden1.data(), 
                              HIDDEN1_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(biasHidden2, hostBiasHidden2.data(), 
                              HIDDEN2_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(biasOutput, hostBiasOutput.data(), 
                              OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
}

NeuralNetwork::~NeuralNetwork() {
    hipFree(weightsInputHidden1);
    hipFree(weightsHidden1Hidden2);
    hipFree(weightsHidden2Output);
    hipFree(biasHidden1);
    hipFree(biasHidden2);
    hipFree(biasOutput);
    hipFree(gradWeightsInputHidden1);
    hipFree(gradWeightsHidden1Hidden2);
    hipFree(gradWeightsHidden2Output);
    hipFree(gradBiasHidden1);
    hipFree(gradBiasHidden2);
    hipFree(gradBiasOutput);
} 